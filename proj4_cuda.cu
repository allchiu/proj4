#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""
#include "proj4.h"

__global__ void computeDiagonalsSumsKernel(unsigned char* input, unsigned char* output, int n, unsigned long target_sum) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n || col >= n) return;

    // Checking Top-left to Bottom-right diagonal 
    unsigned long sum = 0;
    for (int offset = 0; row + offset < n && col + offset < n; offset++) {
        sum += input[(row + offset) * n + (col + offset)];
        if (sum == target_sum) {
            for (int i = 0; i <= offset; i++) {
                output[(row + i) * n + (col + i)] = input[(row + i) * n + (col + i)];
            }
        }
        if (sum > target_sum) {
            break;
        }
    }

    sum = 0;
    for (int offset = 0; row + offset < n && col - offset >= 0; offset++) {
        sum += input[(row + offset) * n + (col - offset)];
        if (sum == target_sum) {
            for (int i = 0; i <= offset; i++) {
                output[(row + i) * n + (col - i)] = input[(row + i) * n + (col - i)];
            }
        }
        if (sum > target_sum) {
            break;
        }
    }
}

extern "C" float diagonalSumsCUDA(grid* input, unsigned long target_sum, grid* output) {
    hipEvent_t start, stop;  
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); 
    output->n = input->n;
    output->p = (unsigned char**)malloc(output->n * sizeof(unsigned char*));
    if (!output->p) {
        printf("Memory allocation failed for output grid\n");
        return 0;
    }
    for (unsigned int i = 0; i < output->n; i++) {
        output->p[i] = (unsigned char*)calloc(output->n, sizeof(unsigned char));
        if (!output->p[i]) {
            printf("Memory allocation failed for output grid row %d\n", i);
            return 0;
        }
    }

    unsigned char* flat_input = (unsigned char*)malloc(input->n * input->n * sizeof(unsigned char));
    unsigned char* flat_output = (unsigned char*)malloc(input->n * input->n * sizeof(unsigned char));

    if (!flat_input || !flat_output) {
        printf("Memory allocation failed for input or output arrays\n");
        return 0;
    }

    for (unsigned int i = 0; i < input->n; i++) {
        for (unsigned int j = 0; j < input->n; j++) {
            flat_input[i * input->n + j] = input->p[i][j];
        }
    }

    unsigned char* d_input, * d_output;
    hipMalloc(&d_input, input->n * input->n * sizeof(unsigned char));
    hipMalloc(&d_output, input->n * input->n * sizeof(unsigned char));

    hipMemcpy(d_input, flat_input, input->n * input->n * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemset(d_output, 0, input->n * input->n * sizeof(unsigned char));

    dim3 blockSize(16, 16);
    dim3 gridSize((input->n + blockSize.x - 1) / blockSize.x,
                  (input->n + blockSize.y - 1) / blockSize.y);            
    computeDiagonalsSumsKernel<<<gridSize, blockSize>>>(d_input, d_output, input->n, target_sum);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    hipMemcpy(flat_output, d_output, input->n * input->n * sizeof(unsigned char), hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < output->n; i++) {
        for (unsigned int j = 0; j < output->n; j++) {
            output->p[i][j] = flat_output[i * output->n + j];  // Fix indexing error here
        }
    }

    free(flat_input);
    free(flat_output);
    hipFree(d_input);
    hipFree(d_output);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds;
}